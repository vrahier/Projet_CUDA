#include "cume_gpu_timer.h"

using namespace cume;

GPUTimer::GPUTimer() {
	hipEventCreate(&t_start);
	hipEventCreate(&t_stop);
}

/**
 * destructor
 */
GPUTimer::~GPUTimer() {
	hipEventDestroy(t_start);
	hipEventDestroy(t_stop);
}

/**
 * start timer
 */
void GPUTimer::start() {
	hipEventRecord(t_start, 0);
}

/**
 * stop timer
 */
void GPUTimer::stop() {
	hipEventRecord(t_stop, 0);
	hipEventSynchronize(t_stop);
}	

/**
 * print timer difference in milliseconds
 */
ostream& GPUTimer::print(ostream& out) {
	float elapsed_time;
	hipEventElapsedTime(&elapsed_time, t_start, t_stop);
	out.setf(ios::fixed);
	out.precision(2);
	out << elapsed_time << "ms";
	return out;
}
