#include "gpu_timer.h"


GPUTimer::GPUTimer() {
	hipEventCreate(&t_start);
	hipEventCreate(&t_stop);
}

GPUTimer::~GPUTimer() {
	hipEventDestroy(t_start);
	hipEventDestroy(t_stop);
}

void GPUTimer::start() {
	hipEventRecord(t_start, NULL);
}

void GPUTimer::stop() {
	hipEventRecord(t_stop, NULL);
	hipEventSynchronize(t_stop);
}

ostream& GPUTimer::print(ostream& out) {
	float milli_seconds = 0.0f;
	hipEventElapsedTime(&milli_seconds, t_start, t_stop);
	out << fixed;
	out << milli_seconds;
	return out;
}

